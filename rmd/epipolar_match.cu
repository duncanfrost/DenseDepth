#include "hip/hip_runtime.h"
// This file is part of REMODE - REgularized MOnocular Depth Estimation.
//
// Copyright (C) 2014 Matia Pizzoli <matia dot pizzoli at gmail dot com>
// Robotics and Perception Group, University of Zurich, Switzerland
// http://rpg.ifi.uzh.ch
//
// REMODE is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// REMODE is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#ifndef RMD_EPIPOLAR_MATCH_CU
#define RMD_EPIPOLAR_MATCH_CU

#include <float.h>
#include <rmd/se3.cuh>
#include <rmd/seed_matrix.cuh>
#include <rmd/mvs_device_data.cuh>
#include <rmd/texture_memory.cuh>
#include <rmd/helper_vector_types.cuh>

namespace rmd
{

__constant__ Size c_img_size;
extern "C" void copyImgSzToConst(rmd::Size *h_img_size)
{
  hipMemcpyToSymbol(HIP_SYMBOL(c_img_size), h_img_size, sizeof(rmd::Size));
}

__global__
void seedEpipolarMatchKernel(
    mvs::DeviceData *dev_ptr,
    SE3<float> T_curr_ref)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x >= c_img_size.width || y >= c_img_size.height)
    return;

  const float xx = x+0.5f;
  const float yy = y+0.5f;

  const int seed_state = tex2D(convergence_tex, xx, yy);
  if( (ConvergenceStates::BORDER    == seed_state) ||
      (ConvergenceStates::CONVERGED == seed_state) ||
      (ConvergenceStates::DIVERGED  == seed_state) )
  {
    return;
  }

  // Retrieve current estimations of depth
  const float mu = tex2D(mu_tex, xx, yy);
  const float sigma = sqrtf(tex2D(sigma_tex, xx, yy));

  const float2 px_ref = make_float2((float)x, (float)y);
  const float3 f_ref = normalize(dev_ptr->cam.cam2world(px_ref));
  const float2 px_mean_curr =
      dev_ptr->cam.world2cam( T_curr_ref * (f_ref * mu) );

  const float2 px_min_curr =
      dev_ptr->cam.world2cam( T_curr_ref * (f_ref * fmaxf( mu - 3.0f*sigma, 0.01f)) );
  const float2 px_max_curr =
      dev_ptr->cam.world2cam( T_curr_ref * (f_ref * ( mu + (3.0f*sigma) ) ) );

  const float2 epi_line = px_max_curr - px_min_curr;
  const float2 epi_dir  = normalize(epi_line);
  const float  half_length = 0.5f * fminf(norm(epi_line), RMD_MAX_EXTENT_EPIPOLAR_SEARCH);
  float2 px_curr, best_px_curr;

  // Retrieve template statistics for NCC matching;
  const float sum_templ = tex2D(sum_templ_tex, xx, yy);
  const float const_templ_denom = tex2D(const_templ_denom_tex, xx, yy);

  // init best match score
  float best_ncc = -1.0f;

  float sum_img;
  float sum_img_sq;
  float sum_img_templ;
  for(float l = -half_length; l <= half_length; l += 0.7f)
  {
    px_curr = px_mean_curr + l*epi_dir;
    if( (px_curr.x >= c_img_size.width - RMD_CORR_PATCH_SIDE)  ||
        (px_curr.y >= c_img_size.height - RMD_CORR_PATCH_SIDE) ||
        (px_curr.x < RMD_CORR_PATCH_SIDE)                    ||
        (px_curr.y < RMD_CORR_PATCH_SIDE) )
    {
      continue;
    }

    sum_img       = 0.0f;
    sum_img_sq    = 0.0f;
    sum_img_templ = 0.0f;

    for(int patch_y=0; patch_y<RMD_CORR_PATCH_SIDE; ++patch_y)
    {
      for(int patch_x=0; patch_x<RMD_CORR_PATCH_SIDE; ++patch_x)
      {
        const float templ = tex2D(
              ref_img_tex,
              px_ref.x+(float)(RMD_CORR_PATCH_OFFSET+patch_x)+0.5f,
              px_ref.y+(float)(RMD_CORR_PATCH_OFFSET+patch_y)+0.5f);
        const float img = tex2D(
              curr_img_tex,
              px_curr.x+(float)(RMD_CORR_PATCH_OFFSET+patch_x)+0.5f,
              px_curr.y+(float)(RMD_CORR_PATCH_OFFSET+patch_y)+0.5f);
        sum_img    += img;
        sum_img_sq += img*img;
        sum_img_templ += img*templ;
      }
    }
    const float ncc_numerator = RMD_CORR_PATCH_AREA*sum_img_templ - sum_img*sum_templ;
    const float ncc_denominator = (RMD_CORR_PATCH_AREA*sum_img_sq - sum_img*sum_img)*const_templ_denom;

    const float ncc = ncc_numerator * rsqrtf(ncc_denominator + FLT_MIN);

    if(ncc > best_ncc)
    {
      best_px_curr = px_curr;
      best_ncc = ncc;
    }
  }
  if(best_ncc < 0.5f)
  {
    dev_ptr->convergence->atXY(x, y) = ConvergenceStates::NO_MATCH;
  }
  else
  {
    dev_ptr->epipolar_matches->atXY(x, y) = best_px_curr;
    dev_ptr->convergence->atXY(x, y) = ConvergenceStates::UPDATE;
  }
}

} // rmd namespace

#endif
