#include "hip/hip_runtime.h"
// This file is part of REMODE - REgularized MOnocular Depth Estimation.
//
// Copyright (C) 2014 Matia Pizzoli <matia dot pizzoli at gmail dot com>
// Robotics and Perception Group, University of Zurich, Switzerland
// http://rpg.ifi.uzh.ch
//
// REMODE is free software: you can redistribute it and/or modify it under the
// terms of the GNU General Public License as published by the Free Software
// Foundation, either version 3 of the License, or any later version.
//
// REMODE is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
// FOR A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this program.  If not, see <http://www.gnu.org/licenses/>.

#ifndef RMD_SEED_UPDATE_CU
#define RMD_SEED_UPDATE_CU

#include <rmd/mvs_device_data.cuh>
#include <rmd/seed_matrix.cuh>
#include <rmd/texture_memory.cuh>

#include "triangulation.cu"

namespace rmd
{

__device__ __forceinline__
float normpdf(
    const float &x,
    const float &mu,
    const float & sigma_sq)
{
  return (expf(-(x-mu)*(x-mu) / (2.0f*sigma_sq))) * rsqrtf(2.0f*M_PI*sigma_sq);
}

__global__
void seedUpdateKernel(
    mvs::DeviceData *dev_ptr,
    SE3<float> T_ref_curr)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if(x >= dev_ptr->width || y >= dev_ptr->height)
    return;

  const float xx = x+0.5f;
  const float yy = y+0.5f;

  // Check convergence status of seed
  if( (ConvergenceStates::CONVERGED == tex2D(convergence_tex, xx, yy)) ||
      (ConvergenceStates::DIVERGED  == tex2D(convergence_tex, xx, yy)) )
    return;

  if( ConvergenceStates::UPDATE == tex2D(convergence_tex, xx, yy) )
  {
    // Bayesian update
    // Retrieve current estimations of parameters
    const float mu = tex2D(mu_tex, xx, yy);
    const float sigma_sq = tex2D(sigma_tex, xx, yy);
    const float a = tex2D(a_tex, xx, yy);
    const float b = tex2D(b_tex, xx, yy);

    // The pixel in reference frame
    const float2 px_ref = make_float2((float)x, (float)y);
    const float3 f_ref = normalize(dev_ptr->cam.cam2world(px_ref));
    // and the epipolar match
    const float2 epipolar_match = tex2D(epipolar_matches_tex, xx, yy);
    const float3 f_epi_match  = normalize(dev_ptr->cam.cam2world(epipolar_match));
    const float3 pt_xyz_ref = triangulatenNonLin(
          f_ref,
          f_epi_match,
          T_ref_curr);
    if(pt_xyz_ref.z < 0.0f)
    {
      return;
    }
    const float depth = norm(pt_xyz_ref);
    //float z = pt_xyz_ref.z;
    float tau = triangulationUncertainty(
          depth,
          f_ref,
          T_ref_curr.getTranslation(),
          dev_ptr->cam.getOnePixAngle());
    const float tau_sq = tau * tau;
    const float s_sq = (tau_sq * sigma_sq) / (tau_sq + sigma_sq);
    const float m    = s_sq * (mu / sigma_sq + depth / tau_sq);
    float c1   = (a / (a+b)) * normpdf(depth, mu, sigma_sq+tau_sq);
    float c2   = (b / (a+b)) * (1.0f / dev_ptr->scene.depth_range);
    const float norm_const = c1 + c2;
    c1 = c1 / norm_const;
    c2 = c2 / norm_const;
    const float f = c1 * ((a + 1.0f) / (a + b + 1.0f)) + c2 *(a / (a + b + 1.0f));
    const float e = c1 * (( (a + 1.0f)*(a + 2.0f)) / ((a + b + 1.0f) * (a + b + 2.0f))) +
        c2 *(a*(a + 1.0f) / ((a + b + 1.0f) * (a + b + 2.0f)));

    if(isnan(c1*m))
    {
      return;
    }

    const float mu_prime = c1 * m + c2 * mu;
    dev_ptr->sigma->atXY(x, y) = c1 *(s_sq + m*m) + c2 * (sigma_sq + mu*mu) - mu_prime*mu_prime;
    dev_ptr->mu->atXY(x, y) = mu_prime;
    const float a_prime = ( e - f ) / ( f - e/f );
    dev_ptr->a->atXY(x, y) = a_prime;
    dev_ptr->b->atXY(x, y) = a_prime * ( 1.0f-f ) / f;
  }

  else if(ConvergenceStates::NO_MATCH == tex2D(convergence_tex, xx, yy))
  { // no match but projection inside the image: penalize the seed
    const float b = tex2D(b_tex, xx, yy) + 1.0f;
    dev_ptr->b->atXY(x, y) = b;
  }
  else if (ConvergenceStates::NOT_VISIBLE == tex2D(convergence_tex, xx, yy))
  { // no match, projection out of the image
  }
}

} // rmd namespace

#endif
