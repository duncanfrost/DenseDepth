#include "hip/hip_runtime.h"
#include "MonoDepthEstimator_CUDA.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace MonoLib;

inline dim3 getBlocksFor2DProcess(int _width,int _height)
{
    return dim3((_width + 16 - 1) / 16, (_height + 16 - 1) / 16);
}

inline dim3 getThreadsFor2DProcess(int _width,int _height)
{
    return dim3(16,16);
}


inline float SumError(float *data, Vector2i imgSize)
{
    float error = 0;
    for (int y = 0; y < imgSize.y; y++)
        for (int x = 0; x < imgSize.x; x++)
        {
            unsigned int index = x + imgSize.x*y;
            error += data[index];
        }

    return error;
}


template <typename T>
__device__ T clamp(T in, T max)
{
    T out = in;
    if (in > max - 1)
        out = max - 1;
    if (in < 0)
        out = 0;

    return out;
}

template <typename T>
__device__ T HuberNorm(T x, T eps)
{
    T absX = abs(x); 

    if (absX <= eps)
        return absX*absX / (2 * eps);
    else
        return absX - eps/2;
}

__global__ void InitVolumeValues_device(float *photo_error,
                                        int *nUpdates,
                                        Vector2i imgSize, int depthSamples)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;

    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    for (int z = 0; z < depthSamples; z++)
    {
        int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
        photo_error[offset] = 0;
        nUpdates[offset] = 0;
    }
}
__global__ void Init2DValues_device(float *g_data, Vector4u *imageData,
                                    Vector2i imgSize, float alpha, float beta)
{

    int id_x = blockIdx.x*blockDim.x+threadIdx.x;
    int id_y = blockIdx.y*blockDim.y+threadIdx.y;
    if (id_x > imgSize.x - 1 || id_y > imgSize.y - 1) return;
    int offset = id_x + id_y * imgSize.x;

    int x_plus = clamp(id_x+1, imgSize.x);
    int x_mid = clamp(id_x, imgSize.x);

    int y_plus = clamp(id_y+1, imgSize.y);
    int y_mid = clamp(id_y, imgSize.y);

    float pixMid = colourToIntensity(imageData[x_mid + imgSize.x * y_mid]);
    float pixXPlus = colourToIntensity(imageData[ x_plus + imgSize.x* y_mid]);
    float pixYPlus = colourToIntensity(imageData[ x_mid + imgSize.x* y_plus]);

    float dIx = pixXPlus - pixMid;
    float dIy = pixYPlus - pixMid;

    float norm = dIx*dIx + dIy*dIy;

    float normb = powf(norm,beta);

    g_data[offset]=expf(-alpha*normb);
}

__global__ void UpdateDQ(float *g_data, float *qx_data,float *qy_data,
                         float *d_data, float *a_data, Vector2i imgSize,
                         float minIDepth, float maxIDepth, float epsilon, float sigma_q,
                         float sigma_d, float theta)
{
    int id_x = blockIdx.x*blockDim.x+threadIdx.x;
    int id_y = blockIdx.y*blockDim.y+threadIdx.y;
    if (id_x > imgSize.x - 1 || id_y > imgSize.y - 1) return;

    int offset = id_x + id_y * imgSize.x;


    int x_plus = clamp(id_x + 1, imgSize.x);
    int y_plus = clamp(id_y + 1, imgSize.y);

    //Gradient from matrix A
    float grad_d_x=d_data[x_plus + imgSize.x * id_y] - d_data[id_x + imgSize.x * id_y];
    float grad_d_y=d_data[id_x + imgSize.x * y_plus] - d_data[id_x + imgSize.x * id_y];

    //This is the weigting based on gradient
    float gx=g_data[id_x + imgSize.x*id_y];
    //	gx = 1;

    float new_qx=(qx_data[offset]+sigma_q*gx*grad_d_x)/(1.+sigma_q*epsilon);
    float new_qy=(qy_data[offset]+sigma_q*gx*grad_d_y)/(1.+sigma_q*epsilon);

    float magn=sqrtf(new_qx*new_qx+new_qy*new_qy);
    if(magn>1)
    {
        new_qx=new_qx/magn;
        new_qy=new_qy/magn;
    }

    qx_data[offset]=new_qx;
    qy_data[offset]=new_qy;

    //Update D
    __syncthreads();

    int x_minus = clamp(id_x-1,imgSize.x);
    int y_minus = clamp(id_y-1,imgSize.y);

    float grad_qx_x=qx_data[id_x + imgSize.x * id_y] - qx_data[x_minus + imgSize.x*id_y];
    float grad_qy_y=qy_data[id_x + imgSize.x * id_y] - qy_data[id_x + imgSize.x*y_minus];


    // TODO: I'm sure this is a mistake. This should be negative, but doesn't work
    float Atransq = 1*(grad_qx_x+grad_qy_y);
    float new_d=(d_data[offset]+sigma_d*(gx*Atransq+1.0f* a_data[offset]/theta))/(1.+sigma_d/theta);;
    if(new_d<minIDepth) new_d=minIDepth;
    if(new_d>maxIDepth) new_d=maxIDepth;

    d_data[offset]=new_d;
}


__global__ void ComputeGradient(float *image_data, Vector2i imgSize,
                                float *gradx_data, float *grady_data)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset = x + y * imgSize.x;

    int x_plus = clamp(x + 1, imgSize.x);
    int y_plus = clamp(y + 1, imgSize.y);

    //Gradient from matrix A
    gradx_data[image_offset]=image_data[x_plus + imgSize.x * y] - image_data[x + imgSize.x * y];
    grady_data[image_offset]=image_data[x + imgSize.x * y_plus] - image_data[x + imgSize.x * y];
}


__global__ void UpdateD(float *d_data, float *divQ_data,
                        float *a_data, float *g_data,
                        float sigma_d, float theta,Vector2i imgSize)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset = x + y * imgSize.x;


    float invTheta = 1 / theta;

    float divQ = divQ_data[image_offset];
    float d_in = d_data[image_offset];
    float g = g_data[image_offset];
    float a = a_data[image_offset];

    float d_out = d_in + sigma_d*(g*divQ + invTheta*a);

    d_out /= (1 + sigma_d*invTheta);

    float norm = max_agnostic(1, abs_agnostic(d_out));

    d_data[image_offset] = d_out / norm;

}

__global__ void UpdateDL1(float *d_data, float *divQ_data,
                          float *a_data, float *p_data,
                          float sigma, float tau, float lambda,
                          Vector2i imgSize)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset = x + y * imgSize.x;


    float divQ = divQ_data[image_offset];
    float d_in = d_data[image_offset];
    float p = p_data[image_offset];

    float d_out = d_in + tau*divQ - lambda*(tau*p);
    d_data[image_offset] = d_out;
}


__global__ void UpdateQ(float *qx_data,float *qy_data,
                        float *dx_data,float *dy_data,
                        float *g_data, 
                        Vector2i imgSize, float sigma_q, float epsilon)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset = x + y * imgSize.x;

    float qx = qx_data[image_offset];
    float qy = qy_data[image_offset];

    float dx = dx_data[image_offset];
    float dy = dy_data[image_offset];

    float g = g_data[image_offset];

    float qx_out = qx + sigma_q * g * dx;
    float qy_out = qy + sigma_q * g * dy;

    qx_out /= (1 + sigma_q * epsilon);
    qy_out /= (1 + sigma_q * epsilon);

    float norm_q = sqrt(qx_out*qx_out + qy_out*qy_out);
    float norm = max_agnostic(1, norm_q);
    qx_out /= norm;
    qy_out /= norm;


    qx_data[image_offset] = qx_out;
    qy_data[image_offset] = qy_out;
}

__global__ void UpdateQL1(float *qx_data,float *qy_data,
                          float *dx_data,float *dy_data,
                          Vector2i imgSize, float sigma_q)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

    int image_offset = x + y * imgSize.x;

    float qx = qx_data[image_offset];
    float qy = qy_data[image_offset];

    float dx = dx_data[image_offset];
    float dy = dy_data[image_offset];

    float qx_out = qx + sigma_q * dx;
    float qy_out = qy + sigma_q * dy;

    float norm_q = sqrt(qx_out*qx_out + qy_out*qy_out);
    float norm = max_agnostic(1, norm_q);
    qx_out /= norm;
    qy_out /= norm;


    qx_data[image_offset] = qx_out;
    qy_data[image_offset] = qy_out;
}

__global__ void UpdatePL1(float *p_data,
                          float *d_data,float *a_data,
                          Vector2i imgSize, float sigma,
                          float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

    int image_offset = x + y * imgSize.x;

    float p = p_data[image_offset];
    float d = d_data[image_offset];
    float a = a_data[image_offset];

    float p_out = p + sigma*lambda*(d - a);

    float norm_p = abs(p_out);
    float norm = max_agnostic(1, norm_p);
    p_out /= norm;

    p_data[image_offset] = p_out;
}


__global__ void ComputeDivQ(float *qx_data,float *qy_data,
                            float *divQ_data, Vector2i imgSize)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset = x + y * imgSize.x;


    float qx_left;
    if (x == 0)
        qx_left = 0;
    else
    {
        int x_minus = x-1;
        int offset_x_minus = x_minus + y * imgSize.x;
        qx_left = qx_data[offset_x_minus];
    }

    float qy_up;
    if (y == 0)
        qy_up = 0;
    else
    {
        int y_minus = y-1;
        int offset_y_minus = x + y_minus * imgSize.x;
        qy_up = qy_data[offset_y_minus];
    }

    float qx_mid = qx_data[image_offset];
    float qy_mid = qy_data[image_offset];

    float divQ = qy_mid - qy_up;
    divQ += qx_mid - qx_left ;

    divQ_data[image_offset] = divQ;
}

__global__ void OptimToDepth_device(float *optim_data, float *depth_data,
                                    float minIDepth, float maxIDepth,
                                    Vector2i imgSize)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset = x + y * imgSize.x;


    float iDepthDiff = maxIDepth - minIDepth;
    float inverseDepth = minIDepth + iDepthDiff * optim_data[image_offset];
    depth_data[image_offset] = 1 / inverseDepth;
}





__global__ void UpdateDQCombined(float *photo_error, float *g_data, float *qx_data,float *qy_data,
                                 float *d_data, float *a_data, Vector2i imgSize,
                                 float minIDepth, float maxIDepth, float epsilon, float sigma_q,
                                 float sigma_d, float depthIncrement, int depthSamples, float theta,
                                 float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

    int image_offset = x + y * imgSize.x;


    int x_plus = clamp(x + 1, imgSize.x);
    int y_plus = clamp(y + 1, imgSize.y);

    //Gradient from matrix A
    float grad_d_x=d_data[x_plus + imgSize.x * y] - d_data[x + imgSize.x * y];
    float grad_d_y=d_data[x + imgSize.x * y_plus] - d_data[x + imgSize.x * y];

    //This is the weigting based on gradient
    float gx=g_data[x + imgSize.x*y];
    //	gx = 1;

    float new_qx=(qx_data[image_offset]+sigma_q*gx*grad_d_x)/(1.+sigma_q*epsilon);
    float new_qy=(qy_data[image_offset]+sigma_q*gx*grad_d_y)/(1.+sigma_q*epsilon);

    float magn=sqrtf(new_qx*new_qx+new_qy*new_qy);
    if(magn>1)
    {
        new_qx=new_qx/magn;
        new_qy=new_qy/magn;
    }

    qx_data[image_offset]=new_qx;
    qy_data[image_offset]=new_qy;



    //Update D
    __syncthreads();

    int x_minus = clamp(x-1,imgSize.x);
    int y_minus = clamp(y-1,imgSize.y);

    float grad_qx_x=qx_data[x + imgSize.x * y] - qx_data[x_minus + imgSize.x*y];
    float grad_qy_y=qy_data[x + imgSize.x * y] - qy_data[x + imgSize.x*y_minus];


    // TODO: I'm sure this is a mistake. This should be negative, but doesn't work
    float Atransq = 1*(grad_qx_x+grad_qy_y);
    float new_d=(d_data[image_offset]+sigma_d*(gx*Atransq+1.0f* a_data[image_offset]/theta))/(1.+sigma_d/theta);;
    if(new_d<minIDepth) new_d=minIDepth;
    if(new_d>maxIDepth) new_d=maxIDepth;

    d_data[image_offset]=new_d;

    //Do photo search
    __syncthreads();

    float minError = 9999.0f;
    int minIdx = 0;


    float increment = 1.0f / depthSamples;

    for (unsigned int z = 0; z < depthSamples; z++)
    {
        int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
        float smoothRes =  d_data[image_offset] - increment*(float)z;
        float error=lambda * photo_error[offset] +
            (smoothRes*smoothRes) / (theta * 2.0f);
        if (error < minError)
        {
            minIdx = z;
            minError = error;
        }
    }

    a_data[image_offset]= increment*(float)minIdx;


}

__global__ void InitSmoothingParameters(float *a_data,float *d_data,float *qx_data,
                                        float *gy_data, Vector2i imgSize)
{
    int id_x = blockIdx.x*blockDim.x+threadIdx.x;
    int id_y = blockIdx.y*blockDim.y+threadIdx.y;
    if (id_x < imgSize.x && id_y < imgSize.y)
    {
        int offset = id_x + id_y * imgSize.x;
        d_data[offset]=a_data[offset];
        qx_data[offset]=0;
        gy_data[offset]=0;
    }
}

__global__ void updatePhotoError2d(Matrix3f R, Vector3f T,
                                   Intrinsics K,
                                   Vector2i imgSize,
                                   float *photo_error,
                                   int *nUpdates,
                                   Vector4u *currImageData,
                                   Vector4u *refImageData, unsigned int depthSamples,
                                   float minIDepth, float depthIncrement)
{

    short int x = blockIdx.x*blockDim.x+threadIdx.x;
    short int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;


    float u = x;
    float v = y;

    Vector3f pointRefUnscaled;
    pointRefUnscaled.x = u * K.fxInv + K.cxInv;
    pointRefUnscaled.y = v * K.fyInv + K.cyInv;
    pointRefUnscaled.z = 1;



    for (unsigned int z = 0; z < depthSamples; z++)
    {
        int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;

        float idepth = minIDepth + z*depthIncrement;
        float depth = 1.0f/idepth;

        Vector3f pointTrack = R * (pointRefUnscaled * depth) + T;
        Vector2f pointTrackImage;
        pointTrackImage.x = K.fx * pointTrack.x / pointTrack.z + K.cx;
        pointTrackImage.y = K.fy * pointTrack.y / pointTrack.z + K.cy;


        //if the voxel is projected in the current image then update photometric error and counter
        if (PointInImage(pointTrackImage, imgSize) && pointTrack[2] > 0)
        {
            //L1 norm of the photometric error

            int x_ref_plus = clamp(x+1, imgSize.x);
            int y_ref_plus = clamp(y+1, imgSize.y);

            float x_curr_plus = clamp(pointTrackImage.x + 1, (float)imgSize.x);
            float y_curr_plus = clamp(pointTrackImage.y + 1, (float)imgSize.y);

            Vector4f photo_current_OR =
                interpolateBilinearVec4(currImageData,pointTrackImage.x,
                                        pointTrackImage.y, imgSize.x);
            Vector4u photo_ref = refImageData[x + imgSize.x*y];


            //Compute gradient for ref
            float pixMid = colourToIntensity(photo_ref);
            float pixXPlus = colourToIntensity(refImageData[ x_ref_plus + imgSize.x* y]);
            float pixYPlus = colourToIntensity(refImageData[ x + imgSize.x* y_ref_plus]);
            float dIx_ref = pixXPlus - pixMid;
            float dIy_ref = pixYPlus - pixMid;


            //Compute gradient for current
            pixMid = colourToIntensity(photo_current_OR);
            pixXPlus = colourToIntensity(
                interpolateBilinearVec4(currImageData,x_curr_plus,
                                        pointTrackImage.y, imgSize.x));
            pixYPlus = colourToIntensity(
                interpolateBilinearVec4(currImageData,pointTrackImage.x,
                                        y_curr_plus, imgSize.x));
            float dIx_curr = pixXPlus - pixMid;
            float dIy_curr = pixYPlus - pixMid;

            float normL1 = PhotoErrorL1(photo_current_OR,photo_ref);
            // float normL1 = PhotoErrorL1Grad(photo_current_OR,photo_ref,
            //                             dIx_ref, dIy_ref,
            // dIx_curr, dIy_curr);

            float oldError = photo_error[offset];
            float obsError = normL1;
            int nUpdate = nUpdates[offset];

            float newError = (nUpdate * oldError + obsError) / (nUpdate + 1);

            photo_error[offset] = newError;
            nUpdates[offset] = nUpdate + 1;
        }
    }
}


__global__ void MinPhotoErrorInit(float *photo_error,float *d_data,
                                  float *a_data, int *minIdx_data,
                                  Vector2i imgSize, int depthSamples)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    float minError = 9999.0f;
    int minIdx = 0;

    float increment = 1.0f / (float)(depthSamples - 1);

    for (unsigned int z = 0; z < depthSamples; z++)
    {
        int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
        float error= photo_error[offset];

        if (error < minError)
        {
            minIdx = z;
            minError = error;
        }
    }

    a_data[image_offset] = increment*(float)minIdx;
    d_data[image_offset] = increment*(float)minIdx;
    minIdx_data[image_offset] = minIdx;
}


__global__ void ComputeCertainty(float *photo_error,
                                 int *minIdx_data,
                                 float *certainty_data,
                                 Vector2i imgSize,
                                 int depthSamples)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;


    float c = 0;
    for (unsigned int z = 0; z < depthSamples-2; z++)
    {
        int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
        float error= photo_error[offset];

        int offset_front = x + y * imgSize.x+ (z+1)* imgSize.x*imgSize.y;
        float error_front = photo_error[offset_front];


        int offset_front2 = x + y * imgSize.x+ (z+2)* imgSize.x*imgSize.y;
        float error_front2 = photo_error[offset_front2];

        float grad = error - error_front;
        float grad_front = error_front2 - error_front;

        float grad2 = grad_front - grad;
        c += grad2;
    }

    float val = expf(-c/3);
    if (val > 1) val = 1;

    certainty_data[image_offset] = val;
}






__global__ void MinPhotoError2d_device(float *photo_error,float *d_data,
                                       float *a_data, int *minIdx_data,
                                       Vector2i imgSize,
                                       int depthSamples, float theta,
                                       float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    float minError = 9999.0f;
    int minIdx = 0;

    float increment = 1.0f / (float)(depthSamples - 1);
    float d_value = d_data[image_offset];

    for (unsigned int z = 0; z < depthSamples; z++)
    {
        float error = GetCombinedError(photo_error, d_value,
                                       x,y,z,theta,lambda,increment,imgSize);

        if (error < minError)
        {
            minIdx = z;
            minError = error;
        }
    }

    a_data[image_offset] = increment*(float)minIdx;
    minIdx_data[image_offset] = minIdx;
}


__global__ void MinErrorNaiveFit_device(float *photo_error,float *d_data,
                                        float *a_data, int *minIdx_data,
                                        Vector2i imgSize,
                                        int depthSamples, float theta,
                                        float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    float minError = 9999.0f;
    int minIdx = 0;

    float increment = 1.0f / (float)(depthSamples - 1);
    float d_value = d_data[image_offset];

    for (unsigned int z = 0; z < depthSamples; z++)
    {
        float error = GetCombinedError(photo_error, d_value,
                                       x,y,z,theta,lambda,increment,imgSize);

        if (error < minError)
        {
            minIdx = z;
            minError = error;
        }
    }

    a_data[image_offset] = increment*(float)minIdx;
    minIdx_data[image_offset] = minIdx;

    if (minIdx == 0 || minIdx == depthSamples-1)
        return;

    float errorPlus = GetCombinedError(photo_error, d_value,
                                       x,y,minIdx+1,theta,lambda,increment,imgSize);
    float errorMinus = GetCombinedError(photo_error, d_value,
                                        x,y,minIdx-1,theta,lambda,increment,imgSize);


    Vector3f errors = Vector3f(errorMinus, minError, errorPlus);
    Vector3f param = GetQuadFit(errors);
    float a = param[0];
    float b = param[1];
    float t_best = -b / (2*a);

    a_data[image_offset] += t_best*increment;
}

__global__ void MinErrorTrueFit_device(float *photo_error,float *d_data,
                                       float *a_data, int *minIdx_data,
                                       float *error_data,
                                       Vector2i imgSize,
                                       int depthSamples, float theta,
                                       float eps, float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    float minError = 9999.0f;
    int minIdx = 0;

    float increment = 1.0f / (float)(depthSamples - 1);
    float d_value = d_data[image_offset];

    for (unsigned int z = 0; z < depthSamples; z++)
    {
        float error = GetCombinedError(photo_error, d_value,
                                       x,y,z,theta,lambda,increment,imgSize);

        if (error < minError)
        {
            minIdx = z;
            minError = error;
        }
    }

    a_data[image_offset] = increment*(float)minIdx;
    minIdx_data[image_offset] = minIdx;

    if (minIdx == 0 || minIdx == depthSamples-1)
        return;

    //These are photo errors, not combined errors
    float errorPlus = GetPhotoError(photo_error,x,y,minIdx+1,imgSize);
    float errorMid = GetPhotoError(photo_error,x,y,minIdx,imgSize);
    float errorMinus = GetPhotoError(photo_error,x,y,minIdx-1,imgSize);
    Vector3f param = GetQuadFit(Vector3f(errorMinus, errorMid, errorPlus));
    float a = param[0];
    float b = param[1];
    float c = param[2];

    float a_depth = a_data[image_offset];
    float u = d_value;
    float m = depthSamples - 1;
    float theta_m_m = (1 / (theta*m*m));
    float nom = (u - a_depth)/(theta*m) - lambda*b;
    float denom = 2*lambda*a + theta_m_m;
    float t_best = nom / denom;

    a_depth = a_depth + (t_best/m);
    a_data[image_offset] = a_depth;

    __syncthreads();

    int x_plus = clamp(x+1, imgSize.x);
    int y_plus = clamp(y+1, imgSize.y);

    float grad_a_x=a_data[x_plus + imgSize.x * y] - a_data[x + imgSize.x * y];
    float grad_a_y=a_data[x + imgSize.x * y_plus] - a_data[x + imgSize.x * y];

    float grad_norm = (grad_a_x*grad_a_x + grad_a_y*grad_a_y);
    float absNorm=sqrtf(grad_norm);
    float TV_Huber = HuberNorm(absNorm, eps);

    float photoError = a*t_best*t_best + b*t_best + c;
    float totalError = TV_Huber + lambda * photoError;
    error_data[image_offset] = totalError;
}


__global__ void ComputeFullError_device(float *d_data, float *error_data,
                                        float *photo_error, int *minIdx_data,
                                        Vector2i imgSize, float eps, float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    int x_plus = clamp(x+1, imgSize.x);
    int y_plus = clamp(y+1, imgSize.y);
    // //Gradient from matrix A

    float grad_d_x=d_data[x_plus + imgSize.x * y] - d_data[x + imgSize.x * y];
    float grad_d_y=d_data[x + imgSize.x * y_plus] - d_data[x + imgSize.x * y];

    float grad_norm = (grad_d_x*grad_d_x + grad_d_y*grad_d_y);
    float absNorm=sqrtf(grad_norm);
    float TV_Huber = HuberNorm(absNorm, eps);


    int z = minIdx_data[image_offset];
    int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
    float photoErrorPix = photo_error[offset];

    float totalError = TV_Huber + lambda * photoErrorPix;

    error_data[image_offset] = totalError;
}

__global__ void ComputeL1Error(float *d_data,
                               float *a_data,
                               float *error_data,
                               Vector2i imgSize,
                               float lambda)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    int x_plus = clamp(x+1, imgSize.x);
    int y_plus = clamp(y+1, imgSize.y);
    // //Gradient from matrix A

    float grad_d_x=d_data[x_plus + imgSize.x * y] - d_data[x + imgSize.x * y];
    float grad_d_y=d_data[x + imgSize.x * y_plus] - d_data[x + imgSize.x * y];

    float grad_norm = (grad_d_x*grad_d_x + grad_d_y*grad_d_y);
    float absNorm=sqrtf(grad_norm);

    float d = d_data[image_offset];
    float a = a_data[image_offset];

    float diff = fabs(d - a);

    float totalError = absNorm + lambda * diff;
    error_data[image_offset] = totalError;
}





MonoDepthEstimator_CUDA::MonoDepthEstimator_CUDA(Vector2i imgSize, Vector4f intrinsics_raw)
    :MonoDepthEstimator(imgSize, intrinsics_raw)
{
}

float MonoDepthEstimator_CUDA::EvaluateGT()
{
    MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;

    size_t totalSize = monoLevel->gtDepth->dataSize;

    monoLevel->depth->UpdateHostFromDevice();
    monoLevel->gtDepth->UpdateHostFromDevice();

    float *obtainedDepth = monoLevel->depth->GetData(MEMORYDEVICE_CPU);
    float *gtDepths = monoLevel->gtDepth->GetData(MEMORYDEVICE_CPU);

    float totalMeasured = 0;
    float diffTotal = 0.0f;
    for (size_t locId = 0; locId < totalSize; locId++)
    {
        float gtDepth = gtDepths[locId];
        if (gtDepth > 0.0f || gtDepth > 3.0f)
        {

            float methodDepth = obtainedDepth[locId];
            if (methodDepth > 3.0f) continue;

            diffTotal += fabs(gtDepths[locId] - methodDepth);

            totalMeasured++;
        }
    }

    return diffTotal / (float)totalMeasured;
}

void MonoDepthEstimator_CUDA::SetRefImage(ORUChar4TSImage *frame)
{
    currDepthFrame->Init();
    currDepthFrame->colorImageData->SetFrom(frame, MEMCPYDIR_CUDA_TO_CPU);

    MonoDepthEstimator::SetRefImage(frame);
}

void MonoDepthEstimator_CUDA::ReinitOptim()
{
    float *photoErrors = optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA);
    int *nUpdates = optimPyramid->nUpdates->GetData(MEMORYDEVICE_CUDA);
    MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;
    Vector2i imgSize = currDepthFrame->colorImageData->noDims;

    dim3 blocks2=getBlocksFor2DProcess(imgSize.x ,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x ,imgSize.y);

    Init2DValues_device<<<blocks2,threadsPerBlock2>>>(optimPyramid->g->GetData(MEMORYDEVICE_CUDA),
                                                      currDepthFrame->colorImageData->GetData(MEMORYDEVICE_CUDA),
                                                      imgSize, tvSettings.edgeAlpha, tvSettings.edgeBeta);

    InitVolumeValues_device<<<blocks2,threadsPerBlock2>>>(photoErrors,nUpdates,
                                                          imgSize, optimPyramid->depthSamples);

    monoLevel->nUpdate= 0;

}

void MonoDepthEstimator_CUDA::UpdatePhotoError(ORUtils::SE3Pose refToTracker,
                                               ORUtils::TimeStampedImage<Vector4u> *frame)
{
    float depthIncrement = (optimPyramid->maxIDepth - optimPyramid->minIDepth) /
        (float)optimPyramid->depthSamples;

    MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;
    Vector2i imgSize = monoLevel->depth->noDims;

    dim3 blocks2=getBlocksFor2DProcess(imgSize.x ,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x ,imgSize.y);

    updatePhotoError2d<<<blocks2,threadsPerBlock2>>>(refToTracker.GetR(),
                                                     refToTracker.GetT(),
                                                     monoLevel->intrinsics,
                                                     imgSize,
                                                     optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
                                                     optimPyramid->nUpdates->GetData(MEMORYDEVICE_CUDA),
                                                     frame->GetData(MEMORYDEVICE_CUDA),
                                                     currDepthFrame->colorImageData->GetData(MEMORYDEVICE_CUDA),
                                                     optimPyramid->depthSamples,
                                                     optimPyramid->minIDepth,
                                                     depthIncrement);
    monoLevel->nUpdate++;

    hipDeviceSynchronize();
}

void MonoDepthEstimator_CUDA::RunTVOptimisation(unsigned int iterations)
{
    InitOptim();
    MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;

    Vector2i imgSize = monoLevel->depth->noDims;
    dim3 blocks2=getBlocksFor2DProcess(imgSize.x,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x, imgSize.y);


    // ComputeCertainty<<<blocks2,threadsPerBlock2>>>(optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
    //                                                optimPyramid->minIndices->GetData(MEMORYDEVICE_CUDA),
    //                                                optimPyramid->g->GetData(MEMORYDEVICE_CUDA),
    //                                                imgSize,
    //                                                optimPyramid->depthSamples);


    float thetaStart = 1;
    float thetaEnd = 1e-4;
    float thetaDiff = thetaStart - thetaEnd;
    float outerError = 0;
    iterations = 300;
    float beta = 0.002;


    optimPyramid->photoErrors->UpdateHostFromDevice();

    double totalError = 0;
    long count = 0;

    for (int y = 0; y < imgSize.y; y++)
        for (int x = 0; x < imgSize.x; x++)
            for (int z = 0; z < optimPyramid->depthSamples; z++)
            {
                int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
                float error = optimPyramid->photoErrors->GetData(MEMORYDEVICE_CPU)[offset];
                totalError += error;

                count += 1;
            }


    totalError /= (double)count;


    // std::cout << "Average error: " << totalError << std::endl;
    // exit(1);
    




    // for (unsigned int i = 0; i < iterations; i++)
    float theta = 0.2;
    
    while (theta > thetaEnd)
    {
        // float theta = thetaStart - ((float)i / (float)(iterations-1))*thetaDiff;

        theta = theta*(1-beta);

        float innerErrorStart = 0;


        float sigma_q = tvSettings.sigma_q;
        float sigma_d = tvSettings.sigma_d;

        ComputeFullError_device<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                              optimPyramid->error->GetData(MEMORYDEVICE_CUDA),
                                                              optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
                                                              optimPyramid->minIndices->GetData(MEMORYDEVICE_CUDA),
                                                              imgSize, tvSettings.epsilon, tvSettings.lambda); 


        optimPyramid->error->UpdateHostFromDevice();
        float lastError = SumError(optimPyramid->error->GetData(MEMORYDEVICE_CPU), imgSize);

        for (unsigned int j = 0; j < 10; j++)
        {

            ComputeGradient<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                          imgSize, 
                                                          optimPyramid->dx->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->dy->GetData(MEMORYDEVICE_CUDA));



            UpdateQ<<<blocks2,threadsPerBlock2>>>(optimPyramid->qx->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->qy->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->dx->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->dy->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->g->GetData(MEMORYDEVICE_CUDA),
                                                  imgSize, sigma_q, tvSettings.epsilon);


            ComputeDivQ<<<blocks2,threadsPerBlock2>>>(optimPyramid->qx->GetData(MEMORYDEVICE_CUDA),
                                                      optimPyramid->qy->GetData(MEMORYDEVICE_CUDA),
                                                      optimPyramid->divQ->GetData(MEMORYDEVICE_CUDA),
                                                      imgSize);



            UpdateD<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->divQ->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->g->GetData(MEMORYDEVICE_CUDA),
                                                  sigma_d, theta, imgSize);


            ComputeFullError_device<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                                  optimPyramid->error->GetData(MEMORYDEVICE_CUDA),
                                                                  optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
                                                                  optimPyramid->minIndices->GetData(MEMORYDEVICE_CUDA),
                                                                  imgSize, tvSettings.epsilon, tvSettings.lambda); 


            optimPyramid->error->UpdateHostFromDevice();
            float error = SumError(optimPyramid->error->GetData(MEMORYDEVICE_CPU), imgSize);

            if (error < lastError)
            {
                sigma_d *= 1.001;
                sigma_q *= 1.001;
            }
            else
            {
                sigma_d *= 0.998;
                sigma_q *= 0.998;
            }
                

            if (j == 0) innerErrorStart = error;

            std::cout << "Theta " << theta
                      << " OuterError " << outerError
                      << " Start error: " << innerErrorStart
                      << " Error: " << error
                      << " Sigma d: " << sigma_d 
                      << " Sigma q: " << sigma_q
                      << std::endl;
        }
                                              


        MinErrorTrueFit_device<<<blocks2,threadsPerBlock2>>>(optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
                                                             optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                             optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                             optimPyramid->minIndices->GetData(MEMORYDEVICE_CUDA),
                                                             optimPyramid->error->GetData(MEMORYDEVICE_CUDA),
                                                             imgSize,
                                                             optimPyramid->depthSamples, theta, tvSettings.epsilon,
                                                             tvSettings.lambda);

        optimPyramid->error->UpdateHostFromDevice();
        outerError = SumError(optimPyramid->error->GetData(MEMORYDEVICE_CPU), imgSize);
    }

    OptimToDepth(false);
}


void MonoDepthEstimator_CUDA::InitOptim()
{
    MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;

    Vector2i imgSize = monoLevel->depth->noDims;
    dim3 blocks2=getBlocksFor2DProcess(imgSize.x,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x ,imgSize.y);

    MinPhotoErrorInit<<<blocks2,threadsPerBlock2>>>(optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
                                                    optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                    optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                    optimPyramid->minIndices->GetData(MEMORYDEVICE_CUDA),
                                                    imgSize, optimPyramid->depthSamples);

    InitSmoothingParameters<<<blocks2,threadsPerBlock2>>>(optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->qx->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->qy->GetData(MEMORYDEVICE_CUDA),
                                                          imgSize);


    OptimToDepth(false);
}


void MonoDepthEstimator_CUDA::SmoothDTAM()
{
    // MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;
    // Vector2i imgSize = monoLevel->depth->noDims;
    //update dual variable q
    // dim3 blocks2=getBlocksFor2DProcess(imgSize.x,imgSize.y);
    // dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x, imgSize.y);

    // for (unsigned int i = 0; i < 1; i++)
    // {
    // UpdateDQ<<<blocks2,threadsPerBlock2>>>(optimPyramid->g->GetData(MEMORYDEVICE_CUDA),
    //                                        optimPyramid->qx->GetData(MEMORYDEVICE_CUDA),
    //                                        optimPyramid->qy->GetData(MEMORYDEVICE_CUDA),
    //                                        optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
    //                                        optimPyramid->a->GetData(MEMORYDEVICE_CUDA), imgSize,
    //                                        optimPyramid->minIDepth,optimPyramid->maxIDepth,
    //                                        tvSettings.epsilon, tvSettings.sigma_q,tvSettings.sigma_d,
    //                                        tvSettings.theta);
    // }
}

void MonoDepthEstimator_CUDA::DisplayPhotoVolume(int x, int y)
{
    optimPyramid->photoErrors->UpdateHostFromDevice();
    Vector2i imgSize = optimPyramid->g->noDims;

    for (unsigned int z = 30; z < 35; z++)
    {
        int offset = x + y * imgSize.x+ z* imgSize.x*imgSize.y;
        std::cout << optimPyramid->photoErrors->GetData(MEMORYDEVICE_CPU)[offset] << std::endl;
    }
}
    

__global__ void ComputeTVError_device(float *d_data, float *error_data, Vector2i
                                      imgSize, float eps)
{
    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
    int image_offset=x + y * imgSize.x;

    int x_plus = clamp(x+1, imgSize.x);
    int y_plus = clamp(y+1, imgSize.y);
    // //Gradient from matrix A

    float grad_d_x=d_data[x_plus + imgSize.x * y] - d_data[x + imgSize.x * y];
    float grad_d_y=d_data[x + imgSize.x * y_plus] - d_data[x + imgSize.x * y];

    float grad_norm = (grad_d_x*grad_d_x + grad_d_y*grad_d_y);
    float absNorm=sqrtf(grad_norm);
    float TV_Huber = HuberNorm(absNorm, eps);

    error_data[image_offset] = TV_Huber;
}



void MonoDepthEstimator_CUDA::MeasureError()
{
    MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;
    Vector2i imgSize = monoLevel->depth->noDims;

    dim3 blocks2=getBlocksFor2DProcess(imgSize.x ,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x ,imgSize.y);
    // ComputeTVError_device<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
    //                                                     optimPyramid->error->GetData(MEMORYDEVICE_CUDA),
    //                                                     imgSize, tvSettings.epsilon); 

    ComputeFullError_device<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->error->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->photoErrors->GetData(MEMORYDEVICE_CUDA),
                                                          optimPyramid->minIndices->GetData(MEMORYDEVICE_CUDA),
                                                          imgSize, tvSettings.epsilon, tvSettings.lambda); 
    optimPyramid->UpdateHostFromDevice();

    float error = 0;
    for (int y = 0; y < imgSize.y; y++)
        for (int x = 0; x < imgSize.x; x++)
        {
            unsigned int index = x + imgSize.x*y;
            error += optimPyramid->error->GetData(MEMORYDEVICE_CPU)[index];
        }
    std::cout << "Error " << error << std::endl;
}


void MonoDepthEstimator_CUDA::OptimToDepth(bool useRawDepth)
{
    float *data;
    if (useRawDepth)
        data = optimPyramid->a->GetData(MEMORYDEVICE_CUDA);
    else
        data = optimPyramid->d->GetData(MEMORYDEVICE_CUDA);

    MonoLib::MonoPyramidLevel *monoLevel = currDepthFrame->dataImage;
    Vector2i imgSize = monoLevel->depth->noDims;

    dim3 blocks2=getBlocksFor2DProcess(imgSize.x ,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x ,imgSize.y);
    OptimToDepth_device<<<blocks2,threadsPerBlock2>>>(data, monoLevel->depth->GetData(MEMORYDEVICE_CUDA),
                                                      optimPyramid->minIDepth,optimPyramid->maxIDepth, imgSize);
}



void MonoDepthEstimator_CUDA::SmoothL1()
{

    //D is the same as u
    //A is the same as f_in
    std::cout << "Here" << std::endl;

    Vector2i imgSize = optimPyramid->d->noDims;
    dim3 blocks2=getBlocksFor2DProcess(imgSize.x,imgSize.y);
    dim3 threadsPerBlock2=getThreadsFor2DProcess(imgSize.x, imgSize.y);

    float L2=1.0;
    float tau=0.00051;
    float sigma=1.0/(L2*tau);
    float lambda = 0.8;

    for (unsigned int j = 0; j < 1200; j++)
    {

        ComputeGradient<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                      imgSize, 
                                                      optimPyramid->dx->GetData(MEMORYDEVICE_CUDA),
                                                      optimPyramid->dy->GetData(MEMORYDEVICE_CUDA));

        UpdateQL1<<<blocks2,threadsPerBlock2>>>(optimPyramid->qx->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->qy->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->dx->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->dy->GetData(MEMORYDEVICE_CUDA),
                                                imgSize, sigma);

        UpdatePL1<<<blocks2,threadsPerBlock2>>>(optimPyramid->p->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                imgSize, sigma, lambda);

        ComputeDivQ<<<blocks2,threadsPerBlock2>>>(optimPyramid->qx->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->qy->GetData(MEMORYDEVICE_CUDA),
                                                  optimPyramid->divQ->GetData(MEMORYDEVICE_CUDA),
                                                  imgSize);

        UpdateDL1<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->divQ->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                optimPyramid->p->GetData(MEMORYDEVICE_CUDA),
                                                sigma, tau, lambda, imgSize);

        ComputeL1Error<<<blocks2,threadsPerBlock2>>>(optimPyramid->d->GetData(MEMORYDEVICE_CUDA),
                                                     optimPyramid->a->GetData(MEMORYDEVICE_CUDA),
                                                     optimPyramid->error->GetData(MEMORYDEVICE_CUDA),
                                                     imgSize, lambda);


        optimPyramid->error->UpdateHostFromDevice();
        float error = SumError(optimPyramid->error->GetData(MEMORYDEVICE_CPU), imgSize);
        std::cout << "Error: " << error << std::endl;

    }
    
    std::cout << "Here2" << std::endl;

}